#include "hip/hip_runtime.h"
#include "winograd.cuh"

// Transformation matrices for F(2x2, 3x3)
__constant__ float G[4][3] = {
    {1.0f, 0.0f, 0.0f}, 
    {0.5f, 0.5f, 0.5f}, 
    {0.5f, -0.5f, 0.5f}, 
    {0.0f, 0.0f, 1.0f}
};

__constant__ float B_T[4][4] = {
    {1.0f, 0.0f, -1.0f, 0.0f}, 
    {0.0f, 1.0f, 1.0f, 0.0f}, 
    {0.0f, -1.0f, 1.0f, 0.0f}, 
    {0.0f, 1.0f, 0.0f, -1.0f}
};

__constant__ float B[4][4] = {
    {1.0f,  0.0f,  0.0f,  0.0f}, 
    {0.0f,  1.0f, -1.0f,  1.0f}, 
    {-1.0f, 1.0f,  1.0f,  0.0f}, 
    {0.0f,  0.0f,  0.0f, -1.0f}
};

__constant__ float A_T[2][4] = {
    {1.0f, 1.0f, 1.0f, 0.0f}, 
    {0.0f, 1.0f, -1.0f, -1.0f}
};

// Kernel to precompute filter transformations
__global__
void filter_transform_kernel(const float* __restrict__ filter,
                             float* __restrict__ U,
                             int K, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_filters = K * C;
    if (idx >= total_filters) return;
    
    int k = idx / C;
    int c = idx % C;
    
    // Get pointer to the 3x3 filter for (k, c)
    const float* g = filter + (k * C + c) * 9;
    
    // Get pointer to output 4x4 transformed filter
    float* u_kc = U + (k * C + c) * 16;
    
    // Filter Transform: U = G * g * G^T
    float temp_g[4][3];
    
    // First step: temp_g = G * g
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 3; ++j) {
            temp_g[i][j] = G[i][0] * g[0 * 3 + j] + G[i][1] * g[1 * 3 + j] + G[i][2] * g[2 * 3 + j];
        }
    }
    
    // Second step: u_kc = temp_g * G^T (manually computed G^T multiplication)
    for (int i = 0; i < 4; ++i) {
        u_kc[i * 4 + 0] = temp_g[i][0];
        u_kc[i * 4 + 1] = 0.5f * (temp_g[i][0] + temp_g[i][1] + temp_g[i][2]);
        u_kc[i * 4 + 2] = 0.5f * (temp_g[i][0] - temp_g[i][1] + temp_g[i][2]);
        u_kc[i * 4 + 3] = temp_g[i][2];
    }
}

// Fused kernel for Winograd convolution F(2x2, 3x3) using precomputed filter transforms
__global__
void winograd_conv_kernel(const float* __restrict__ image,
                          const float* __restrict__ filter,
                          float* __restrict__ output,
                          int N, int C, int H, int W, int K, int outH, int outW) {
    // Optimized 3D thread mapping: x=tile_x, y=tile_y, z=batch*output_channel
    int tile_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tile_y = blockIdx.y * blockDim.y + threadIdx.y;
    int nk_idx = blockIdx.z * blockDim.z + threadIdx.z;
    
    int tiles_x = (outW + 1) / 2;
    int tiles_y = (outH + 1) / 2;
    
    // Check bounds
    if (tile_x >= tiles_x || tile_y >= tiles_y || nk_idx >= N * K) return;
    
    // Decompose nk_idx into batch and output channel indices
    int k = nk_idx % K;
    int n = nk_idx / K;

    // Optimized: Use single accumulator array instead of m[4][4]
    float accumulator[16] = {0.0f};

    // Loop over input channels
    for (int c = 0; c < C; ++c) {
        // --- Load Precomputed Filter Transform ---
        const float* u_kc = filter + (k * C + c) * 16;
        
        // --- Image Transform (optimized to use less registers) ---
        int h_start = tile_y * 2;
        int w_start = tile_x * 2;
        
        // Optimized: Reuse temp array for both intermediate steps
        float temp[16];
        
        // Step 1: Load input data and apply B_T transform
        // temp = B_T * d
        for (int i = 0; i < 4; ++i) {
            for (int j = 0; j < 4; ++j) {
                temp[i * 4 + j] = 
                    B_T[i][0] * image[(n * C + c) * H * W + (h_start + 0) * W + (w_start + j)] +
                    B_T[i][1] * image[(n * C + c) * H * W + (h_start + 1) * W + (w_start + j)] +
                    B_T[i][2] * image[(n * C + c) * H * W + (h_start + 2) * W + (w_start + j)] +
                    B_T[i][3] * image[(n * C + c) * H * W + (h_start + 3) * W + (w_start + j)];
            }
        }
        
        // Step 2: Apply B transform and compute element-wise product
        // v = temp * B, then accumulate m += u * v
        for (int i = 0; i < 4; ++i) {
            for (int j = 0; j < 4; ++j) {
                float v_val = 
                    temp[i * 4 + 0] * B[0][j] +
                    temp[i * 4 + 1] * B[1][j] +
                    temp[i * 4 + 2] * B[2][j] +
                    temp[i * 4 + 3] * B[3][j];
                
                accumulator[i * 4 + j] += u_kc[i * 4 + j] * v_val;
            }
        }
    }

    // --- Output Transform (optimized to use minimal registers) ---
    // Compute Y = A_T * accumulator * A
    // Step 1: temp = A_T * accumulator
    float temp_out[8]; // 2x4 result
    for (int i = 0; i < 2; ++i) {
        for (int j = 0; j < 4; ++j) {
            temp_out[i * 4 + j] = 
                A_T[i][0] * accumulator[0 * 4 + j] +
                A_T[i][1] * accumulator[1 * 4 + j] +
                A_T[i][2] * accumulator[2 * 4 + j] +
                A_T[i][3] * accumulator[3 * 4 + j];
        }
    }
    
    // Step 2: Compute final output and write directly
    for (int i = 0; i < 2; ++i) {
        for (int j = 0; j < 2; ++j) {
            float Y_val;
            if (j == 0) {
                Y_val = temp_out[i * 4 + 0] + temp_out[i * 4 + 1] + temp_out[i * 4 + 2];
            } else {
                Y_val = temp_out[i * 4 + 1] - temp_out[i * 4 + 2] - temp_out[i * 4 + 3];
            }
            
            int h = tile_y * 2 + i;
            int w = tile_x * 2 + j;
            if (h < outH && w < outW) {
                output[((n * K + k) * outH + h) * outW + w] = Y_val;
            }
        }
    }
}

void winograd_conv(thrust::device_vector<float>& image,
                   thrust::device_vector<float>& filter, 
                   thrust::device_vector<float>& out,
                   thrust::device_vector<float>& U,
                   thrust::device_vector<float>& V, 
                   thrust::device_vector<float>& M,
                   int H, int W, int C, int K, int N) {
    const int outH = H - 2;
    const int outW = W - 2;
    
    // Step 1: Precompute filter transformations
    const int threads_per_block_filter = 256;
    int total_filters = K * C;
    int grid_size_filter = (total_filters + threads_per_block_filter - 1) / threads_per_block_filter;
    
    filter_transform_kernel<<<grid_size_filter, threads_per_block_filter>>>(
        filter.data().get(), U.data().get(), K, C
    );
    
    // Step 2: Optimized 3D blocking for better memory access pattern
    int tiles_x = (outW + 1) / 2;  // Number of tiles in X direction
    int tiles_y = (outH + 1) / 2;  // Number of tiles in Y direction
    int total_nk = N * K;          // Total batch * output_channel combinations
    
    // Choose block dimensions that work well with new mapping
    // blockDim.x = tile_x, blockDim.y = tile_y, blockDim.z = batch*output_channel
    dim3 blockDim(8, 8, 8);  // 512 threads per block, good for occupancy
    
    // Calculate grid dimensions to cover all (tile_x, tile_y, N*K) combinations
    dim3 gridDim(
        (tiles_x + blockDim.x - 1) / blockDim.x,  // Tile X dimension
        (tiles_y + blockDim.y - 1) / blockDim.y,  // Tile Y dimension  
        (total_nk + blockDim.z - 1) / blockDim.z  // Batch * Output channel dimension
    );

    winograd_conv_kernel<<<gridDim, blockDim>>>(
        image.data().get(), U.data().get(), out.data().get(),
        N, C, H, W, K, outH, outW
    );

    hipDeviceSynchronize();
}